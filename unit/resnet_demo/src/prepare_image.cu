
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void RGBToTensorKernel(const unsigned char *in, float *out, const int w, const int h) {
  // screw it, get someone in the know to explain this
  int pixel_x = threadIdx.x + blockIdx.x * blockDim.x;
  int pixel_y = threadIdx.y + blockIdx.y * blockDim.y;
  const int source_step = w * 3;

  const int plane_size = w * h;
  const int source_pos = source_step * pixel_y + pixel_x * 3;

  const int dest_pos_r = w * pixel_y + pixel_x;
  const int dest_pos_g = dest_pos_r + plane_size;
  const int dest_pos_b = dest_pos_r + plane_size * 2;

  unsigned char in_r = in[source_pos];
  unsigned char in_g = in[source_pos + 1];
  unsigned char in_b = in[source_pos + 2];

  out[dest_pos_r] = (float)in_r / 255.0f;
  out[dest_pos_g] = (float)in_g / 255.0f;
  out[dest_pos_b] = (float)in_b / 255.0f;
  // normalize
  out[dest_pos_r] = (out[dest_pos_r] - 0.485) / 0.229;
  out[dest_pos_g] = (out[dest_pos_g] - 0.456) / 0.224;
  out[dest_pos_b] = (out[dest_pos_b] - 0.406) / 0.225;
}

__host__ void RGBToTensor(const unsigned char *in, float *out, const int w, const int h) {

  constexpr int BLOCK_SIZE = 8;
  dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid_dim((w - 1) / BLOCK_SIZE + 1, (h - 1) / BLOCK_SIZE + 1, 1);

  RGBToTensorKernel<<<grid_dim, block_dim>>>(in, out, w, h);
}